// This program computes matrix multiplication using shared memory tiling
// @author: Zain Tariq


#include <hip/hip_runtime.h>
#include <algorithm>
#include <cassert>
#include <cstdlib>
#include <functional>
#include <iostream>
#include <fstream>
#include <vector>

using std::cout;
using std::generate;
using std::vector;

using namespace std;

// Pull out matrix and shared memory tile size 
const int N = 1 << 11;
const int SHMEM_SIZE = 1 << 10;

__global__ void matrixMul(const int *a, const int *b, int *c) {
  // Compute each thread's global row and column index
  int row = blockIdx.y * blockDim.y + threadIdx.y;
  int col = blockIdx.x * blockDim.x + threadIdx.x;

  // Statically allocated shared memory
  __shared__ int s_a[SHMEM_SIZE];
  __shared__ int s_b[SHMEM_SIZE];

  // Accumulate in temporary variable
  int tmp = 0;
  // Sweep tile across matrix
  for (int i = 0; i < N; i += blockDim.x) {
    // Load in elements for this tile
    s_a[threadIdx.y * blockDim.x + threadIdx.x] = a[row * N + i + threadIdx.x];
    s_b[threadIdx.y * blockDim.x + threadIdx.x] =
        b[i * N + threadIdx.y * N + col];

    // Wait for both tiles to be loaded in before doing computation
    __syncthreads();

    // Do matrix multiplication on the small matrix
    for (int j = 0; j < blockDim.x; j++) {
      tmp +=
          s_a[threadIdx.y * blockDim.x + j] * s_b[j * blockDim.x + threadIdx.x];
    }

    // Wait for all threads to finish using current tiles before loading in new
    // ones
    __syncthreads();
  }

  // Write back results
  c[row * N + col] = tmp;

}

int main() {
  // Size (in bytes) of matrix
  size_t bytes = N * N * sizeof(int);

  // Host vectors
  vector<int> h_a(N * N);
  vector<int> h_b(N * N);
  vector<int> h_c(N * N);

  // Initialize matrices
  generate(h_a.begin(), h_a.end(), []() { return rand() % 100; });
  generate(h_b.begin(), h_b.end(), []() { return rand() % 100; });

  // Allocate device memory
  int *d_a, *d_b, *d_c;
  hipMalloc(&d_a, bytes);
  hipMalloc(&d_b, bytes);
  hipMalloc(&d_c, bytes);

  float gpu_elapsed_time_ms;

  // some events to count the execution time
  hipEvent_t start, stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);

  // start to count execution time of GPU version
  hipEventRecord(start, 0);

  // Copy data to the device
  hipMemcpy(d_a, h_a.data(), bytes, hipMemcpyHostToDevice);
  hipMemcpy(d_b, h_b.data(), bytes, hipMemcpyHostToDevice);

  // Threads per CTA dimension
  int THREADS = 256;

  // Blocks per grid dimension (assumes THREADS divides N evenly)
  int BLOCKS = 64;

  // Use dim3 structs for block  and grid dimensions
  dim3 threads(THREADS, THREADS);
  dim3 blocks(BLOCKS, BLOCKS);

  // Launch kernel
  matrixMul <<<blocks, threads>>> (d_a, d_b, d_c);

  // Copy back to the host
  hipMemcpy(h_c.data(), d_c, bytes, hipMemcpyDeviceToHost);

  // time counting terminate
  hipEventRecord(stop, 0);
  hipEventSynchronize(stop);
  // compute time elapse on GPU computing
  hipEventElapsedTime(&gpu_elapsed_time_ms, start, stop);

  printf("Num Threads = %d\nNum Blocks = %d\n",THREADS,BLOCKS);
  printf("Time elapsed on matrix multiplication of %d x %d on GPU: %f ms.\n", N, N,gpu_elapsed_time_ms);  
  cout << "COMPLETED SUCCESSFULLY\n";

  // Free memory on device
  hipFree(d_a);
  hipFree(d_b);
  hipFree(d_c);

  return 0;
}
